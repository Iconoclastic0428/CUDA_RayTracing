#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <fstream>
#include <hiprand/hiprand_kernel.h>

#include "ray.h"
#include "vec3.h"
#include "hittable.h"
#include "hittable_list.h"
#include "sphere.h"
#include "camera.h"
#include "material.h"
#include "rect.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__device__ vec3 color(const ray& r,
    hittable** world,
    hiprandState* state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.f, 1.f, 1.f);

    for (int i = 0; i < 50; ++i) {
        hit_record rec;
        if (!((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec))) {
            return vec3(0.f, 0.f, 0.f);
        }
        ray scattered;
        vec3 attenuation;
        vec3 emitted = rec.mat_ptr->emitted(0., 0., rec.p);
        if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, state)) {
            cur_attenuation *= attenuation;
            cur_ray = scattered;
        }
        else {
            cur_attenuation = emitted * cur_attenuation;
            return cur_attenuation;
        }
    }
    return vec3(0.f, 0.f, 0.f);
}

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* fb, int max_x, int max_y, int ns, camera** cam, hittable** world, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hittable** d_list, hittable** d_world, camera** d_camera, int nx, int ny, hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0, -1000.0, -1), 1000,
            new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a + RND, 0.2, b + RND);
                if (choose_mat < 0.8f) {
                    auto center2 = center + vec3(0, RND * RND, 0);
                    d_list[i++] = new moving_sphere(center, center2, 0.f, 1.f, 0.2,
                        new lambertian(vec3(RND * RND, RND * RND, RND * RND)));
                }
                else if (choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                        new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world = new hittable_list(d_list, 22 * 22 + 1 + 3);

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0; (lookfrom - lookat).length();
        float aperture = 0.1;
        *d_camera = new camera(lookfrom,
            lookat,
            vec3(0, 1, 0),
            30.0,
            float(nx) / float(ny),
            aperture,
            dist_to_focus,
            0.f,
            1.f);
    }
}

__global__ void free_world(hittable** d_list, hittable** d_world, camera** d_cam) {
    for (int i = 0; i < 8; ++i) {
        delete* (d_list + i);
    }
    delete *(d_world);
    delete* (d_cam);
}

__global__ void simple_light(hittable** d_list, hittable** d_world, camera** d_cam, int nx, int ny) {

    d_list[0] = new rectangle_xz(-10, 10, -10, 10, 0, new lambertian(vec3(0.5, 0.5, 0.5)));
    d_list[1] = new sphere(vec3(0, 2, 0), 2, new lambertian(vec3(0.4, 0.2, 0.1)));
    d_list[2] = new rectangle_xy(3, 5, 1, 3, -2, new diffuse_light(vec3(4, 4, 4)));
    d_list[3] = new sphere(vec3(0, 7, 0), 2, new diffuse_light(vec3(4, 4, 4)));
    *d_world = new hittable_list(d_list, 4);
    *d_cam = new camera(vec3(26, 3, 6), vec3(0, 2, 0), vec3(0, 1, 0), 30.f, float(nx) / float(ny), 0., 10., 0.f, 0.f);
}

__global__ void cornell_box(hittable** d_list, hittable** d_world, camera** d_cam, int nx, int ny) {

    d_list[0] = new rectangle_yz(0, 555, 0, 555, 555, new lambertian(vec3(.12, .45, .15)));
    d_list[1] = new rectangle_yz(0, 555, 0, 555, 0, new lambertian(vec3(.65, .05, .05)));
    d_list[2] = new rectangle_xz(213, 343, 227, 332, 554, new diffuse_light(vec3(15, 15, 15)));
    d_list[3] = new rectangle_xz(0, 555, 0, 555, 0, new lambertian(vec3(0.73, 0.73, 0.73)));
    d_list[4] = new rectangle_xz(0, 555, 0, 555, 555, new lambertian(vec3(0.73, 0.73, 0.73)));
    d_list[5] = new rectangle_xy(0, 555, 0, 555, 555, new lambertian(vec3(0.73, 0.73, 0.73)));
    d_list[6] = new box(vec3(130, 0, 65), vec3(295, 165, 230), new lambertian(vec3(0.73, 0.73, 0.73)));
    d_list[7] = new box(vec3(265, 0, 295), vec3(430, 330, 460), new lambertian(vec3(0.73, 0.73, 0.73)));
    *d_world = new hittable_list(d_list, 8);
    *d_cam = new camera(vec3(278, 278, -800), vec3(278, 278, 0), vec3(0, 1, 0), 40.f, float(nx) / float(ny), 0.f, 10.f, 0.f, 0.f);

}

int main() {
    int nx = 600;
    int ny = 600;
    int tx = 8;
    int ty = 8;
    int ns = 400;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    hiprandState* d_rand_state;
    checkCudaErrors(hipMallocManaged((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));


    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> > (nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    // allocate FB
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));
    hittable** d_list;
    checkCudaErrors(hipMallocManaged((void**)&d_list, 8 * sizeof(hittable*)));
    hittable** d_world;
    checkCudaErrors(hipMallocManaged((void**)&d_world, sizeof(hittable*)));
    camera** d_camera;
    checkCudaErrors(hipMallocManaged((void**)&d_camera, sizeof(camera*)));
    cornell_box << <1, 1 >> > (d_list, d_world, d_camera, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    
    clock_t start, stop;
    start = clock();
    // Render our buffer
    render << <blocks, threads >> > (fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::ofstream image("image.ppm");
    // Output FB as Image
    image << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            image << ir << " " << ig << " " << ib << "\n";
        }
    }
    image.close();

    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(fb));
    hipDeviceReset();

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";
}